
#include <hip/hip_runtime.h>
#include<stdio.h>
#define TBP 256

__global__ void hello_world()
{
	printf("Hello World! My threadId is %d\n",threadIdx.x);
	__syncthreads();
}

int main()
{
	hello_world<<<1,TBP>>>();
	hipDeviceSynchronize();
	return 0;
}

